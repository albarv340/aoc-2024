
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

//  nvcc -o gpu day17-2.cu && ./gpu

__managed__ long long start_A = 190300000000000;
__managed__ long long end_A = 19040000000000; // Too high, around 8^15 * 5 + 15000000000000, which is when the last digits start to look good, but down a couple tens of billions


#define LENGTH 16

__managed__ long long default_B = 0, default_C = 0;
__managed__ int program[LENGTH] = {2, 4, 1, 2, 7, 5, 0, 3, 4, 7, 1, 7, 5, 5, 3, 0};

__device__ long long get_combo(long long num, long long A, long long B, long long C)
{
    if (num <= 3)
        return num;
    if (num == 4)
        return A;
    if (num == 5)
        return B;
    if (num == 6)
        return C;
    return -1;
}

__device__ bool areArraysEqual(int arr1[], int arr2[], int size)
{
    for (int i = 0; i < size; ++i)
    {
        if (arr1[i] != arr2[i])
        {
            return false;
        }
    }
    return true;
}

__global__ void testValidity()
{
    long long index = (long long)(blockIdx.y * gridDim.x + blockIdx.x) * (long long)(blockDim.x * blockDim.y) + (long long)(threadIdx.y * blockDim.x + threadIdx.x);
    long long A, B, C;
    long long i = start_A + index;
    if (i % 1000000000 == 0)
        printf("Reached: %lld, %d, %d, %d, %d, %d, %d, %lld\n", i, threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, index);
    A = i;
    B = default_B;
    C = default_C;

    int out[LENGTH];
    int out_i{0};
    int ip{0};
    while (ip < LENGTH)
    {
        int op = program[ip];
        int in = program[ip + 1];
        switch (op)
        {
        case 0:
            A = A >> get_combo(in, A, B, C);
            break;
        case 1:
            B = B ^ in;
            break;
        case 2:
            B = get_combo(in, A, B, C) % 8;
            break;
        case 3:
            if (A != 0)
            {
                ip = in;
                continue;
            }
            break;
        case 4:
            B = B ^ C;
            break;
        case 5:
            out[out_i++] = get_combo(in, A, B, C) % 8;
            break;
        case 6:
            B = A >> get_combo(in, A, B, C);
            break;
        case 7:
            C = A >> get_combo(in, A, B, C);
            break;
        default:
            break;
        }

        ip += 2;
    }

    if (areArraysEqual(out, program, LENGTH))
    {
        printf("Solution: %lld\n", i);
    }
}

int main()
{

    dim3 dimBlock(32, 32);
    dim3 dimGrid(10000, 10000);

    testValidity<<<dimGrid, dimBlock>>>();
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Kernel Launch Error: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }

    return 0;
}
